#include "hip/hip_runtime.h"
#include "nbody_simulation.hpp"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

__global__ void calculate(glm::vec3* accelerations, body* bodies, int num_bodies) {
    const float G = 6.67430f*std::pow(10.0f, -11);
    const float epsilon = 0.0000001f;

    int body_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (body_idx >= num_bodies) return;
    for (unsigned j = 0; j < num_bodies; j++) {
        if (j == body_idx) continue;
        glm::vec3 r(bodies[j].pos-bodies[body_idx].pos);
        glm::vec3 accel(glm::normalize(r)*G*bodies[j].mass/(glm::dot(r, r)+epsilon));
        accelerations[body_idx] += accel;
    }
}

void nbody_simulation::naive_gpu_calculcate_accelerations() {
    int num_bodies = bodies.size();

    glm::vec3 *gpu_accelerations;
    hipMallocManaged(&gpu_accelerations, num_bodies * sizeof(glm::vec3));
    std::memset(gpu_accelerations, 0, num_bodies * sizeof(glm::vec3));

    body *gpu_bodies;
    hipMallocManaged(&gpu_bodies, num_bodies * sizeof(body));
    for (int i = 0; i < num_bodies; i++) {
        gpu_bodies[i] = bodies[i];
    }

    dim3 dim_block(256, 1, 1);
    int grid_size = (num_bodies+dim_block.x-1)/dim_block.x;
    dim3 dim_grid(grid_size, 1, 1);

    calculate<<<dim_grid, dim_block>>>(gpu_accelerations, gpu_bodies, num_bodies);
    hipDeviceSynchronize();

    accelerations.assign(num_bodies, glm::vec3());
    for (int i = 0; i < num_bodies; i++) {
        accelerations[i] = gpu_accelerations[i];
    }

    hipFree(gpu_accelerations);
    hipFree(gpu_bodies);
}

void nbody_simulation::barnes_hut_gpu_calculcate_accelerations() {
    rect root_rect {get_bounding_rect()};

    std::vector<int> keys(bodies.size());
    for (unsigned i = 0; i < bodies.size(); i++) {
        const auto &constrained_pos = convert_xy(root_rect, bodies[i].pos.x, bodies[i].pos.y);
        keys[i] = get_key(constrained_pos.first, constrained_pos.second);
    }

    thrust::device_vector<int> device_keys = keys;
    thrust::device_vector<body> device_bodies = bodies;

    thrust::sort_by_key(device_keys.begin(), device_keys.begin() + device_keys.size(), device_bodies.begin());

    thrust::host_vector<int> sorted_keys = device_keys;
    for (auto val : sorted_keys) {
        std::cout << val << std::endl;
    }
}