#include "hip/hip_runtime.h"
#include "nbody_simulation.hpp"

#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/memory.h>
#include <thrust/device_new.h>

#include <chrono>

__global__ void calculate(glm::vec3* accelerations, body* bodies, int num_bodies) {
    const float G = 6.67430f*std::pow(10.0f, -11);
    const float epsilon = 0.0000001f;

    int body_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (body_idx >= num_bodies) return;
    for (unsigned j = 0; j < num_bodies; j++) {
        if (j == body_idx) continue;
        glm::vec3 r(bodies[j].pos-bodies[body_idx].pos);
        glm::vec3 accel(glm::normalize(r)*G*bodies[j].mass/(glm::dot(r, r)+epsilon));
        accelerations[body_idx] += accel;
    }
}

void nbody_simulation::naive_gpu_calculate_accelerations() {
    int num_bodies = bodies.size();

    glm::vec3 *gpu_accelerations;
    hipMallocManaged(&gpu_accelerations, num_bodies * sizeof(glm::vec3));
    std::memset(gpu_accelerations, 0, num_bodies * sizeof(glm::vec3));

    body *gpu_bodies;
    hipMallocManaged(&gpu_bodies, num_bodies * sizeof(body));
    for (int i = 0; i < num_bodies; i++) {
        gpu_bodies[i] = bodies[i];
    }

    dim3 dim_block(256, 1, 1);
    int grid_size = (num_bodies+dim_block.x-1)/dim_block.x;
    dim3 dim_grid(grid_size, 1, 1);

    calculate<<<dim_grid, dim_block>>>(gpu_accelerations, gpu_bodies, num_bodies);
    hipDeviceSynchronize();

    accelerations.assign(num_bodies, glm::vec3());
    for (int i = 0; i < num_bodies; i++) {
        accelerations[i] = gpu_accelerations[i];
    }

    hipFree(gpu_accelerations);
    hipFree(gpu_bodies);
}


//device_node device_add_layer(device_node &tree, const thrust::device_vector<uint64_t> &keys) {
//    constexpr uint64_t all_ones_last = 0b1111111111111111111111111111111111111111111111111111111111111110ull;
//    constexpr uint64_t three_mask = 0b111ull;
//
//    int stage = tree.size();
//    int sec_last = stage-1;
//
//    thrust::inclusive_scan(tree.);
//}

//__device__ void device_add_layer(device_node* tree, uint64_t *keys, int curr_size) {
//    constexpr uint64_t all_ones = 0b1111111111111111111111111111111111111111111111111111111111111111ull;
//    constexpr uint64_t all_ones_last = 0b1111111111111111111111111111111111111111111111111111111111111110ull;
//    constexpr uint64_t three_mask = 0b111ull;
//
//    int stage = curr_size;
//    int sec_last = stage - 1;
//
//    int prev_num_indices = tree[sec_last].n;
//
//    thrust::inclusive_scan(thrust::device, tree[sec_last].children_index, tree[sec_last].children_index+prev_num_indices, tree[sec_last].children_index);
//    int new_num_indices = tree[sec_last].children_index[prev_num_indices-1]*8;
//    make_device_node(tree[stage], new_num_indices);
//    thrust::counting_iterator<int> start_idx(0);
//    thrust::counting_iterator<int> end_idx = start_idx + prev_num_indices*8;
//
//    // MUST BE COPY!
//    thrust::for_each(thrust::device, start_idx, end_idx, [=](int data) {
//        int idx = data >> 3;
//        int extension_idx = data & 0b111;
//
//        int first_idx = tree[sec_last].start_index[idx];
//        int last_idx = tree[sec_last].end_index[idx];
//        if (first_idx == -1) return;
//
//        uint64_t first_value = keys[first_idx];
//        uint64_t base_mask = all_ones_last << (63 - sec_last*3);
//        uint64_t base_value = first_value & base_mask;
//        int shift_amt = 61 - sec_last*3;
//
//        uint64_t extension = static_cast<uint64_t>(extension_idx) << shift_amt;
//        uint64_t extension_mask = three_mask << shift_amt;
//
//        uint64_t search_value = base_value | extension;
//        uint64_t search_mask = base_mask | extension_mask;
//
//        uint64_t extension_upper = all_ones >> ((sec_last+1)*3);
//        uint64_t search_value_upper = search_value | extension_upper;
//
//        auto found_start = thrust::lower_bound(thrust::device, keys+first_idx, keys+last_idx+1, search_value);
//        if (found_start == keys+last_idx+1 || (((*found_start) & search_mask) != search_value)) return;
//
//        auto found_end = thrust::upper_bound(thrust::device, keys+first_idx, keys+last_idx+1, search_value_upper);
//
//        int found_start_idx = found_start - keys;
//        int found_end_idx = found_end - keys;
//
//        int curr_stage_node_idx = (tree[sec_last].children_index[idx]-1) * 8 + extension_idx;
//        tree[stage].start_index[curr_stage_node_idx] = found_start_idx;
//        tree[stage].end_index[curr_stage_node_idx] = found_end_idx-1;
//        tree[stage].children_index[curr_stage_node_idx] = 1;
//
////        auto found = std::lower_bound(keys+first_idx, keys+last_idx+1, search_value);
//    });
//}

//__global__ void barnes_hut_gpu_function(glm::vec3* accelerations, body* bodies, uint64_t *keys, int n, int tree_depth, float barnes_hut_factor) {
////    thrust::sort_by_key(thrust::seq, keys, keys + n, bodies);
////    thrust::device_ptr<uint64_t> keys_ptr(keys);
////    thrust::sort(thrust::device, keys_ptr, keys_ptr + n);
////    glm::vec3 tot = {0, 0, 0};
////    uint64_t key_tot = 0;
////    for (int i = 0; i < n; i++) {
////        tot += bodies[i].pos;
////        key_tot += keys[i];
////    }
////    test_func<<<1, 1>>>(bodies, keys, n);
//
//    auto *com_mass_ = new nbody_simulation::com_mass[n+1];
//    nbody_simulation::com_mass tmp{{0, 0, 0}, 0};
//    com_mass_[0] = tmp;
//    thrust::transform(thrust::device, bodies, bodies + n, com_mass_+1, device_extract_com());
//    thrust::inclusive_scan(thrust::device, com_mass_, com_mass_+n+1, com_mass_, device_combine_com());
//
//    auto *tree = new device_node[tree_depth+1];
//
//    make_device_node(tree[0], 1);
//    tree[0].start_index[0] = 0;
//    tree[0].end_index[0] = n-1;
//    tree[0].children_index[0] = 1;
//
//    for (int i = 0; i < tree_depth; i++) {
//        device_add_layer(tree, keys, i+1);
//    }
//
//    for (int i = 0; i < tree_depth; i++) {
//        free_device_node(tree[i]);
//    }
//
//    delete[] tree;
//    delete[] com_mass_;
//}

struct device_node{
    int *start_index;
    int *end_index;
    int *children_index;
    int n;
};

struct device_extract_com {
    __device__ nbody_simulation::com_mass operator()(const body &body_) const {
        return {body_.pos, body_.mass};
    }
};

struct device_combine_com {
    __device__ nbody_simulation::com_mass operator()(const nbody_simulation::com_mass &a, const nbody_simulation::com_mass &b) {
        return {(a.com * a.mass + b.com * b.mass) / (a.mass + b.mass), a.mass + b.mass};
    }
};

__device__ void make_device_node(device_node &d_node, int n) {
    d_node.start_index = new int[n];
    memset(d_node.start_index, -1, n*sizeof(int));

    d_node.end_index = new int[n];
    memset(d_node.end_index, -1, n*sizeof(int));

    d_node.children_index = new int[n];
    memset(d_node.children_index, 0, n*sizeof(int));

    d_node.n = n;
//    printf("Made layer with %d items\n", n);
}

__host__ void initialize_device_node_from_host(device_node &d_node, int n) {
    hipMalloc(&d_node.start_index, n * sizeof(int));
    hipMemset(d_node.start_index, 0xFF, n * sizeof(int)); // this will set all the ints to -1

    hipMalloc(&d_node.end_index, n * sizeof(int));
    hipMemset(d_node.end_index, 0xFF, n * sizeof(int)); // this will set all the ints to -1

    hipMalloc(&d_node.children_index, n * sizeof(int));
    hipMemset(d_node.children_index, 0, n * sizeof(int)); // this will set all the ints to 0

    d_node.n = n;
//    printf("Made layer with %d items\n", n);
}

__host__ void free_device_node_from_host(device_node &d_node) {
    hipFree(d_node.start_index);
    hipFree(d_node.end_index);
    hipFree(d_node.children_index);
}

__device__ void free_device_node(device_node &d_node) {
    delete[] d_node.start_index;
    delete[] d_node.end_index;
    delete[] d_node.children_index;
}

struct device_initialize_tree {
    template <typename Tuple>
    __device__ void operator() (Tuple t) {
        device_node &layer = thrust::get<0>(t);
        int num_bodies = thrust::get<1>(t);
        make_device_node(layer, 1);
        layer.start_index[0] = 0;
        layer.end_index[0] = num_bodies-1;
        layer.children_index[0] = 1;
    }
};

struct device_initialize_layer {
    template <typename Tuple>
    __device__ void operator() (Tuple t) {
        device_node &layer = thrust::get<0>(t);
        int new_num_indices = thrust::get<1>(t);
        make_device_node(layer, new_num_indices*8);
//        printf("hi!\n");
    }
};

__device__ void print_binary(uint64_t val, char *out) {
    uint64_t mask = 0b1;
    mask <<= 63;
    for (int i = 0; i < 64; i++) {
//        printf((val & mask) ? "1":"0");
        out[i] = (val & mask) ? '1':'0';
        mask>>=1;
    }
}

struct device_compute_layer {
    template <class Tuple>
    __device__ void operator() (Tuple t) {
        device_node *tree = thrust::get<0>(t);
        uint64_t *keys = thrust::get<1>(t);
        int data = thrust::get<2>(t);
        int stage = thrust::get<3>(t);
        int sec_last = stage - 1;

        constexpr uint64_t all_ones = 0b1111111111111111111111111111111111111111111111111111111111111111ull;
        constexpr uint64_t all_ones_last = 0b1111111111111111111111111111111111111111111111111111111111111110ull;
        constexpr uint64_t three_mask = 0b111ull;

        int idx = data >> 3;
        int extension_idx = data & 0b111;

//        printf("%d %d %d\n", stage, idx, extension_idx);

        int first_idx = tree[sec_last].start_index[idx];
        int last_idx = tree[sec_last].end_index[idx];
//        printf("i: %d %d %d %d\n", first_idx, last_idx, idx, extension_idx);
        if (first_idx == -1) return;
//
//        if (first_idx >= 100001 || last_idx >= 100001 || first_idx < 0 || last_idx < 0) {
//            printf("yo, %d %d %d %d\n", first_idx, last_idx, sec_last, idx);
//        }

        uint64_t first_value = keys[first_idx];
        uint64_t base_mask = all_ones_last << (63 - sec_last*3);
        uint64_t base_value = first_value & base_mask;
        int shift_amt = 61 - sec_last*3;

        uint64_t extension = static_cast<uint64_t>(extension_idx) << shift_amt;
        uint64_t extension_mask = three_mask << shift_amt;

        uint64_t search_value = base_value | extension;
        uint64_t search_mask = base_mask | extension_mask;

        uint64_t extension_upper = all_ones >> ((sec_last+1)*3);
        uint64_t search_value_upper = search_value | extension_upper;

        auto found_start = thrust::lower_bound(thrust::seq, keys+first_idx, keys+last_idx+1, search_value);
        int found_start_idx = found_start - keys;
        //        bool should_return = found_start == keys+last_idx+1 || (((*found_start) & search_mask) != search_value);
//        if (should_return) {
////            tree[0].children_index[0] = 0;
////            return;
//        }
//        if (found_start == keys+last_idx+1 || (((*found_start) & search_mask) != search_value)) return;
        if (found_start == keys + last_idx + 1) {
//            printf("early: %d\n", found_start_idx);
            return;
        };
        if ((((*found_start) & search_mask) != search_value)) {
//            char one[65], two[65], three[65], four[65];
//            memset(one, 0, 65);
//            memset(two, 0, 65);
//            memset(three, 0, 65);
//            memset(four, 0, 65);
//            print_binary(*found_start, one);
//            print_binary(search_mask, two);
//            print_binary(((*found_start) & search_mask), three);
//            print_binary(search_value, four);
//            printf("%d, %s %s %s %s\n", found_start_idx, one, two, three, four);
//            printf("%ull %ull %ull %ull\n", *found_start, search_mask, ((*found_start) & search_mask), search_value);
return;
        }

        auto found_end = thrust::upper_bound(thrust::seq, keys+first_idx, keys+last_idx+1, search_value_upper);
        int found_end_idx = found_end - keys;

//        printf("stage: %d, %d %d\n", stage, found_start_idx, found_end_idx);

        int curr_stage_node_idx = (tree[sec_last].children_index[idx]-1) * 8 + extension_idx;
//        printf("curr_idx: %d\n", curr_stage_node_idx);
//        printf("%d\n", tree[stage].start_index[curr_stage_node_idx]);
        tree[stage].start_index[curr_stage_node_idx] = found_start_idx;
        tree[stage].end_index[curr_stage_node_idx] = found_end_idx-1;
        tree[stage].children_index[curr_stage_node_idx] = 1;

//        tree[stage].start_index[curr_stage_node_idx] = first_idx;
//        tree[stage].end_index[curr_stage_node_idx] = last_idx;


//        printf("currn: %d, prevd: %d, idx: %d, stage: %d\n", tree[stage].n, tree[stage-1].n, idx, stage);
    }
};

struct device_calc_acceleration {
    template <class Tuple>
    __device__ void operator() (Tuple t) {
        // tree_iterator, bodies_iterator, com_mass_iterator, accelerations_iterator, idxes_iterator, focus_idx_iterator, size_iterator, barnes_hut_iterator, tree_depth_iterator
        device_node *tree = thrust::get<0>(t);
        body *bodies = thrust::get<1>(t);
        nbody_simulation::com_mass *com_mass_ = thrust::get<2>(t);
        glm::vec3 *accelerations = thrust::get<3>(t);
        int *idxes = thrust::get<4>(t);
        int focus_idx = thrust::get<5>(t);
        float rect_size = thrust::get<6>(t);
        float barnes_hut_factor = thrust::get<7>(t);
        int tree_depth = thrust::get<8>(t);

//        printf("%d, %f, %f, %d\n", focus_idx, rect_size, barnes_hut_factor, tree_depth);

        const float G = 6.67430f*std::pow(10.0f, -11.0f);
        const float epsilon = 0.000001f;

        const body &curr_body = bodies[focus_idx];
        int write_idx = idxes[focus_idx];

        auto com_combiner = device_combine_com();

        auto query_com = [&com_mass_, &com_combiner](int l, int r) {
            nbody_simulation::com_mass tmp = com_mass_[l];
            tmp.mass = -tmp.mass;
            return com_combiner(com_mass_[r+1], tmp);
        };

        struct stack_item{
            float size;
            int layer;
            int node_idx;
        };
        stack_item stack[20];
        int stack_size = 0;

        auto push_item = [&stack, &stack_size](stack_item item) {
            stack[stack_size] = item;
            stack_size++;
        };

        auto pop_item = [&stack, &stack_size]() {
            stack_size--;
            return stack[stack_size];
        };

        push_item({rect_size, 0, 0});

        glm::vec3 tmp_total_acceleration{0, 0, 0};
        while (stack_size > 0) {
            auto item = pop_item();
            int start = tree[item.layer].start_index[item.node_idx];
            int end = tree[item.layer].end_index[item.node_idx];

            nbody_simulation::com_mass current_com_mass = query_com(start, end);
            float s2 = item.size * item.size;
            glm::vec3 diff = curr_body.pos - glm::vec3(current_com_mass.com);
            float d2 = glm::dot(diff, diff) + epsilon;

            bool inside = start <= focus_idx && focus_idx <= end;
            if (end-start+1 == (int)inside) continue;

            if (s2/d2 < barnes_hut_factor) {
                nbody_simulation::com_mass other_com_mass = current_com_mass;
                if (inside) {
                    nbody_simulation::com_mass tmp{curr_body.pos, -curr_body.mass};
                    other_com_mass = com_combiner(other_com_mass, tmp);
                }

                glm::vec3 r(glm::vec3(other_com_mass.com)-curr_body.pos);
                glm::vec3 accel(glm::normalize(r)*G*(float)other_com_mass.mass/(glm::dot(r, r) + epsilon));
                tmp_total_acceleration += accel;
            } else {
                if (item.layer == tree_depth) {
                    for (int i = tree[item.layer].start_index[item.node_idx]; i <= tree[item.layer].end_index[item.node_idx]; i++) {
                        if (i == focus_idx) continue;
                        glm::vec3 r(bodies[i].pos-curr_body.pos);
                        glm::vec3 accel(glm::normalize(r)*G*bodies[i].mass/(glm::dot(r, r) + epsilon ));
                        tmp_total_acceleration += accel;
                    }
                } else {
                    int children_base_idx = (tree[item.layer].children_index[item.node_idx]-1)*8;

                    for (int rel_child = 0; rel_child<8; rel_child++) {
                        int child_idx = children_base_idx + rel_child;

                        if (tree[item.layer+1].start_index[child_idx] != -1) {
                            push_item({item.size/2, item.layer+1, child_idx});
                        }
                    }
                }

            }
        }

        accelerations[write_idx] = tmp_total_acceleration;
    }
};

struct device_functor_free_node {
    __device__ void operator() (device_node &d_node) {
        free_device_node(d_node);
    }
};

//struct device_prefix_sum_layer {
//    __device__ void operator() (int idx) {
//        device_node &layer = thrust::get<0>(t);
//        int num_bodies = thrust::get<1>(t);
//        make_device_node(layer, 1);
//        layer.start_index[0] = 0;
//        layer.end_index[0] = num_bodies-1;
//        layer.children_index[0] = 1;
//    }
//};

//__global__ void debug_kernel(int* dat, int n) {
//    for (int i = 0; i < n; i++) {
//        printf("%d: %d\n", i, dat[i]);
//    }
//}

void nbody_simulation::barnes_hut_gpu_calculate_accelerations() {
    rect3d root_rect {get_bounding_rect()};

//    size_t free, total;
//    printf("\n");
//    hipMemGetInfo(&free,&total);
//    printf("%d MB free of total %d MB\n",free/1024/1024,total/1024/1024);

    static bool called = false;
    if (!called) {
        hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ll*1024ll*1024ll*1ll); //set limit to 1gb
    }
    called = true;

    int num_bodies = bodies.size();

    if (num_bodies < 2) {
        return;
    }

    std::vector<uint64_t> keys(num_bodies);
    std::vector<int> idxes(num_bodies);
    for (int i = 0; i < num_bodies; i++) {
        const auto &constrained_pos = convert_xyz(root_rect, bodies[i].pos);
        keys[i] = get_key(constrained_pos.x, constrained_pos.y, constrained_pos.z);
        idxes[i] = i;
    }

    thrust::device_vector<uint64_t> device_keys = keys;
    thrust::device_vector<uint64_t> device_keys_copy = device_keys;

    thrust::device_vector<body> device_bodies = bodies;
    thrust::sort_by_key(device_keys.begin(), device_keys.end(), device_bodies.begin());

    thrust::device_vector<int> device_idxes = idxes;
    thrust::sort_by_key(device_keys_copy.begin(), device_keys_copy.end(), device_idxes.begin());

    thrust::device_vector<com_mass> com_mass_(num_bodies+1, com_mass{{0, 0, 0}, 0});
    thrust::transform(device_bodies.begin(), device_bodies.end(), com_mass_.begin()+1, device_extract_com());
    thrust::inclusive_scan(com_mass_.begin(), com_mass_.end(), com_mass_.begin(), device_combine_com());

    thrust::device_vector<device_node> tree(tree_depth+1);
    thrust::constant_iterator<int> num_bodies_iterator(num_bodies-1);

//    thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(tree.begin(), num_bodies_iterator)),
//                     1,
//                     device_initialize_tree());

    device_node host_layer = tree[0];
    initialize_device_node_from_host(host_layer, 1);
    int *init_mem = (int*) malloc(sizeof(int)*3);
    init_mem[0] = 0;
    init_mem[1] = num_bodies-1;
    init_mem[2] = 1;
    hipMemcpy(host_layer.start_index, init_mem+0, sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(host_layer.end_index, init_mem+1, sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(host_layer.children_index, init_mem+2, sizeof (int), hipMemcpyHostToDevice);
    free(init_mem);
    tree[0] = host_layer;

    device_node *tree_ptr = thrust::raw_pointer_cast(tree.data());
    auto tree_iterator = thrust::make_constant_iterator(tree_ptr);

    for (int i = 0; i < tree_depth; i++) {
        auto start = std::chrono::system_clock::now();
        int stage = i + 1;
        int sec_last = i;

        int prev_n = tree[sec_last].operator device_node().n;
        int *children_index = tree[sec_last].operator device_node().children_index;

        auto device_children_index = thrust::device_pointer_cast(children_index);

        thrust::inclusive_scan(device_children_index, device_children_index+prev_n, device_children_index);
//        int prev_layer_size;
//        hipMemcpy(&prev_layer_size, children_index+prev_n);
//        thrust::device_ptr<int> last_children_idx = thrust::device_new<int>(1);
//        thrust::transform(thrust::device, children_index+prev_n-1, children_index+prev_n, last_children_idx, );
        hipEvent_t start_cuda2, stop_cuda2;
        hipEventCreate(&start_cuda2);
        hipEventCreate(&stop_cuda2);
        hipEventRecord(start_cuda2);
//        thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(tree.begin()+stage,device_children_index+prev_n-1)),
//                           1,
//                           device_initialize_layer());
//        int last_children_value = -1;
        int *last_children_value_ptr = (int*) malloc(sizeof(int));
//        *last_children_value_ptr = -2;
//        std::cout << "children: " << *last_children_value_ptr << std::endl;
//        std::cout << "pre: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;
        hipMemcpy(last_children_value_ptr, children_index+prev_n-1, sizeof(int), hipMemcpyDeviceToHost);
//        std::cout << "in: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;
//        std::cout << "in: " << hipGetErrorName(hipPeekAtLastError()) << std::endl;
//        std::cout << "children: " << *last_children_value_ptr << std::endl;
        device_node host_curr_layer = tree[stage];
        initialize_device_node_from_host(host_curr_layer, (*last_children_value_ptr)*8);
        tree[stage] = host_curr_layer;
        hipEventRecord(stop_cuda2);
        hipEventSynchronize(stop_cuda2);
        float milliseconds2 = 0;
        hipEventElapsedTime(&milliseconds2, start_cuda2, stop_cuda2);
//        std::cout << "1: " << milliseconds2 << " ms" << std::endl;

        int last_children_value = *last_children_value_ptr;
        free(last_children_value_ptr);

//        std::cout << "pre: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;
//
//        int *last_children_value_ptr = new int;
//        *last_children_value_ptr = -2;
//        std::cout << "children: " << *last_children_value_ptr << std::endl;
//        hipMemcpy(last_children_value_ptr, children_index+prev_n-1, sizeof(int), hipMemcpyDeviceToHost);
//        std::cout << "in: " << hipGetErrorString(hipPeekAtLastError()) << std::endl;
//        std::cout << "in: " << hipGetErrorName(hipPeekAtLastError()) << std::endl;
//        std::cout << "children: " << *last_children_value_ptr << std::endl;

//        int last_children_value = *last_children_value_ptr;
//        int last_children_value = prev_n;

//        auto curr_stage = tree.begin() + stage;
//        auto prev_stage = tree.begin() + sec_last;
//        std::cout << "paralleled: " << (prev_n * 8) << " " << (last_children_value*8) << std::endl;

        last_children_value = prev_n;

        uint64_t *keys_ptr = thrust::raw_pointer_cast(device_keys.data());
        auto keys_iterator = thrust::make_constant_iterator(keys_ptr);
        auto idx = thrust::make_counting_iterator(0);
        auto stage_iterator = thrust::make_constant_iterator(stage);
        auto zipped_start = thrust::make_zip_iterator(thrust::make_tuple(tree_iterator, keys_iterator, idx, stage_iterator));
        auto zipped_end = thrust::make_zip_iterator(thrust::make_tuple(tree_iterator+last_children_value*8, keys_iterator+last_children_value*8, idx+last_children_value*8, stage_iterator+last_children_value*8));
        hipEvent_t start_cuda, stop_cuda;
        hipEventCreate(&start_cuda);
        hipEventCreate(&stop_cuda);
        hipEventRecord(start_cuda);
        thrust::for_each(thrust::device, zipped_start, zipped_end, device_compute_layer());
        hipEventRecord(stop_cuda);
        hipEventSynchronize(stop_cuda);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start_cuda, stop_cuda);
//        std::cout << "2: " << milliseconds << " ms" << std::endl;

//        std::cout << "paralleled: " << (prev_n * 8) << " " << (last_children_value*8) << std::endl;
//        std::cout << "e" << std::endl;

//        auto prev_tree_level_it = tree.begin() + sec_last;
//        auto transformed_iterator = thrust::make_transform_iterator(prev_tree_level, );

//        // combine into one, fix function above, considering extending from unary_function, can this even be done?
//        thrust::constant_iterator<thrust::device_vector<device_node>::iterator> tree_iterator(tree.begin());
//        thrust::constant_iterator<int> sec_last_iterator(sec_last);
//        auto transform_start_iterator = thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(tree_iterator, sec_last_iterator)), device_children_iterator());
//        auto transform_end_iterator = thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(tree_iterator, sec_last_iterator)), device_children_iterator());
//        thrust::inclusive_scan(thrust::device, transform_start_iterator, transform_end_iterator, transform_start_iterator);

//        thrust::inclusive_scan(thrust::device, );
        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
//        std::cout << "time for layer " << i << ": " << elapsed_seconds.count() << " s\n";
    }

    body *bodies_ptr = thrust::raw_pointer_cast(device_bodies.data());
    auto bodies_iterator = thrust::make_constant_iterator(bodies_ptr);

    com_mass *com_masses = thrust::raw_pointer_cast(com_mass_.data());
    auto com_mass_iterator = thrust::make_constant_iterator(com_masses);

    accelerations.assign(bodies.size(), glm::vec3());
    thrust::device_vector<glm::vec3> device_accelerations(num_bodies, {0, 0, 0});

    glm::vec3 *accelerations_ptr = thrust::raw_pointer_cast(device_accelerations.data());
    auto accelerations_iterator = thrust::make_constant_iterator(accelerations_ptr);

    int *idxes_ptr = thrust::raw_pointer_cast(device_idxes.data());
    auto idxes_iterator = thrust::make_constant_iterator(idxes_ptr);

    auto focus_idx_iterator = thrust::make_counting_iterator(0);

    auto size_iterator = thrust::make_constant_iterator(root_rect.sx);

    auto barnes_hut_iterator = thrust::make_constant_iterator(barnes_hut_factor);

    auto tree_depth_iterator = thrust::make_constant_iterator(tree_depth);

//    std::cout << "depth " << tree_depth << std::endl;

    auto start_calculation_iter = thrust::make_zip_iterator(thrust::make_tuple(tree_iterator, bodies_iterator, com_mass_iterator, accelerations_iterator, idxes_iterator, focus_idx_iterator, size_iterator, barnes_hut_iterator, tree_depth_iterator));
    auto end_calculation_iter = thrust::make_zip_iterator(thrust::make_tuple(tree_iterator+num_bodies, bodies_iterator+num_bodies, com_mass_iterator+num_bodies, accelerations_iterator+num_bodies, idxes_iterator+num_bodies, focus_idx_iterator+num_bodies, size_iterator+num_bodies, barnes_hut_iterator+num_bodies, tree_depth_iterator+num_bodies));
    thrust::for_each(thrust::device, start_calculation_iter, end_calculation_iter, device_calc_acceleration());


//    for (int i = 0; i < num_bodies; i++) {
//        int orig_idx = std::get<2>(bodies_with_keys[i]);
//        accelerations[orig_idx] = calc_acceleration(tree, sorted_bodies, com_mass_, i, root_rect);
//    }

    thrust::host_vector<glm::vec3> host_accelerations;
    host_accelerations = device_accelerations;
    for (int i = 0; i < num_bodies; i++) {
        accelerations[i] = host_accelerations[i];
    }

    for (int i = 0; i < tree_depth+1; i++) {
        device_node layer = tree[i];
        free_device_node_from_host(layer);
    }

//    exit(0);

//    thrust::for_each(tree.begin(), tree.end(), device_functor_free_node());
//    exit(0);

//    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(tree.begin(), num_bodies_iterator)), thrust::make_zip_iterator(thrust::make_tuple(tree.begin()+1, num_bodies_iterator+1)), device_initialize_tree);



//    thrust::host_vector<uint64_t> host_keys = device_keys;



//    std::sort(keys.begin(), keys.end());

//    body *device_bodies;
//    hipHostMalloc(&device_bodies, num_bodies * sizeof(body));
//
//    hipMemcpy(device_bodies, bodies.data(), num_bodies, hipMemcpyHostToDevice);
//
////    std::cout << hipGetErrorName(err) << std::endl;
//
//    uint64_t *device_keys;
//    hipHostMalloc(&device_keys, num_bodies * sizeof(uint64_t));
//
//    hipMemcpy(device_keys, keys.data(), num_bodies, hipMemcpyHostToDevice);
//
//    glm::vec3 *device_accelerations;
//    hipHostMalloc(&device_accelerations, num_bodies * sizeof(glm::vec3));
//
////    for (int i = 0; i < num_bodies; i++) {
////        device_bodies[i] = bodies[i];
////        device_keys[i] = keys[i];
////        device_accelerations[i] = {0, 0, 0};
////    }
//
////    std::cout << "bruh" << std::endl;
////    thrust::sort_by_key(thrust::device, device_keys, device_keys + num_bodies, device_bodies);
////    std::cout << "eet" << std::endl;
////    std::cout << "here" << std::endl;
//    thrust::device_ptr<uint64_t> device_keys_ptr(device_keys);
//    thrust::device_ptr<body> device_bodies_ptr(device_bodies);
//    thrust::sort_by_key(device_keys_ptr, device_keys_ptr + num_bodies, device_bodies_ptr);
////    std::cout << "oop" << std::endl;
//    barnes_hut_gpu_function<<<1, 1>>>(device_accelerations, device_bodies, device_keys, num_bodies, tree_depth, barnes_hut_factor);
//    hipDeviceSynchronize();
////    std::cout << "beep" << std::endl;
//
//    auto kernel_err = hipGetLastError();
//
////    std::cout << "kernel: " << hipGetErrorName(kernel_err) << std::endl;
//
//    hipHostFree(device_bodies);
//    hipHostFree(device_keys);
//    hipHostFree(device_accelerations);

//    exit(0);

//    thrust::device_vector<uint64_t> device_keys = keys;
//    thrust::device_vector<body> device_bodies = bodies;
//
//    thrust::sort_by_key(device_keys.begin(), device_keys.begin() + num_bodies, device_bodies.begin());
//    thrust::device_vector<com_mass> com_mass_(num_bodies+1, com_mass{{0, 0, 0}, 0});
//    thrust::transform(device_bodies.begin(), device_bodies.end(), com_mass_.begin() + 1, device_extract_com());
//    thrust::inclusive_scan(com_mass_.begin(), com_mass_.end(), com_mass_.begin(), device_combine_com());
//
//    thrust::device_vector<device_node> tree;
//    tree.push_back(device_node{thrust::device_vector<int>(1, -1), thrust::device_vector<int>(1, -1),thrust::device_vector<int>(1, 0)});
//
//    for (int i = 0; i < tree_depth; i++) {
//        tree.push_back(device_add_layer(tree.back().operator device_node(), device_keys));
//    }
}