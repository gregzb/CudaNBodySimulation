#include "hip/hip_runtime.h"
#include "nbody_simulation.hpp"

#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/memory.h>
#include <thrust/device_new.h>

__global__ void calculate(glm::vec3* accelerations, body* bodies, int num_bodies) {
    const float G = 6.67430f*std::pow(10.0f, -11);
    const float epsilon = 0.0000001f;

    int body_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (body_idx >= num_bodies) return;
    for (unsigned j = 0; j < num_bodies; j++) {
        if (j == body_idx) continue;
        glm::vec3 r(bodies[j].pos-bodies[body_idx].pos);
        glm::vec3 accel(glm::normalize(r)*G*bodies[j].mass/(glm::dot(r, r)+epsilon));
        accelerations[body_idx] += accel;
    }
}

void nbody_simulation::naive_gpu_calculate_accelerations() {
    int num_bodies = bodies.size();

    glm::vec3 *gpu_accelerations;
    hipMallocManaged(&gpu_accelerations, num_bodies * sizeof(glm::vec3));
    std::memset(gpu_accelerations, 0, num_bodies * sizeof(glm::vec3));

    body *gpu_bodies;
    hipMallocManaged(&gpu_bodies, num_bodies * sizeof(body));
    for (int i = 0; i < num_bodies; i++) {
        gpu_bodies[i] = bodies[i];
    }

    dim3 dim_block(256, 1, 1);
    int grid_size = (num_bodies+dim_block.x-1)/dim_block.x;
    dim3 dim_grid(grid_size, 1, 1);

    calculate<<<dim_grid, dim_block>>>(gpu_accelerations, gpu_bodies, num_bodies);
    hipDeviceSynchronize();

    accelerations.assign(num_bodies, glm::vec3());
    for (int i = 0; i < num_bodies; i++) {
        accelerations[i] = gpu_accelerations[i];
    }

    hipFree(gpu_accelerations);
    hipFree(gpu_bodies);
}


//device_node device_add_layer(device_node &tree, const thrust::device_vector<uint64_t> &keys) {
//    constexpr uint64_t all_ones_last = 0b1111111111111111111111111111111111111111111111111111111111111110ull;
//    constexpr uint64_t three_mask = 0b111ull;
//
//    int stage = tree.size();
//    int sec_last = stage-1;
//
//    thrust::inclusive_scan(tree.);
//}

//__device__ void device_add_layer(device_node* tree, uint64_t *keys, int curr_size) {
//    constexpr uint64_t all_ones = 0b1111111111111111111111111111111111111111111111111111111111111111ull;
//    constexpr uint64_t all_ones_last = 0b1111111111111111111111111111111111111111111111111111111111111110ull;
//    constexpr uint64_t three_mask = 0b111ull;
//
//    int stage = curr_size;
//    int sec_last = stage - 1;
//
//    int prev_num_indices = tree[sec_last].n;
//
//    thrust::inclusive_scan(thrust::device, tree[sec_last].children_index, tree[sec_last].children_index+prev_num_indices, tree[sec_last].children_index);
//    int new_num_indices = tree[sec_last].children_index[prev_num_indices-1]*8;
//    make_device_node(tree[stage], new_num_indices);
//    thrust::counting_iterator<int> start_idx(0);
//    thrust::counting_iterator<int> end_idx = start_idx + prev_num_indices*8;
//
//    // MUST BE COPY!
//    thrust::for_each(thrust::device, start_idx, end_idx, [=](int data) {
//        int idx = data >> 3;
//        int extension_idx = data & 0b111;
//
//        int first_idx = tree[sec_last].start_index[idx];
//        int last_idx = tree[sec_last].end_index[idx];
//        if (first_idx == -1) return;
//
//        uint64_t first_value = keys[first_idx];
//        uint64_t base_mask = all_ones_last << (63 - sec_last*3);
//        uint64_t base_value = first_value & base_mask;
//        int shift_amt = 61 - sec_last*3;
//
//        uint64_t extension = static_cast<uint64_t>(extension_idx) << shift_amt;
//        uint64_t extension_mask = three_mask << shift_amt;
//
//        uint64_t search_value = base_value | extension;
//        uint64_t search_mask = base_mask | extension_mask;
//
//        uint64_t extension_upper = all_ones >> ((sec_last+1)*3);
//        uint64_t search_value_upper = search_value | extension_upper;
//
//        auto found_start = thrust::lower_bound(thrust::device, keys+first_idx, keys+last_idx+1, search_value);
//        if (found_start == keys+last_idx+1 || (((*found_start) & search_mask) != search_value)) return;
//
//        auto found_end = thrust::upper_bound(thrust::device, keys+first_idx, keys+last_idx+1, search_value_upper);
//
//        int found_start_idx = found_start - keys;
//        int found_end_idx = found_end - keys;
//
//        int curr_stage_node_idx = (tree[sec_last].children_index[idx]-1) * 8 + extension_idx;
//        tree[stage].start_index[curr_stage_node_idx] = found_start_idx;
//        tree[stage].end_index[curr_stage_node_idx] = found_end_idx-1;
//        tree[stage].children_index[curr_stage_node_idx] = 1;
//
////        auto found = std::lower_bound(keys+first_idx, keys+last_idx+1, search_value);
//    });
//}

//__global__ void barnes_hut_gpu_function(glm::vec3* accelerations, body* bodies, uint64_t *keys, int n, int tree_depth, float barnes_hut_factor) {
////    thrust::sort_by_key(thrust::seq, keys, keys + n, bodies);
////    thrust::device_ptr<uint64_t> keys_ptr(keys);
////    thrust::sort(thrust::device, keys_ptr, keys_ptr + n);
////    glm::vec3 tot = {0, 0, 0};
////    uint64_t key_tot = 0;
////    for (int i = 0; i < n; i++) {
////        tot += bodies[i].pos;
////        key_tot += keys[i];
////    }
////    test_func<<<1, 1>>>(bodies, keys, n);
//
//    auto *com_mass_ = new nbody_simulation::com_mass[n+1];
//    nbody_simulation::com_mass tmp{{0, 0, 0}, 0};
//    com_mass_[0] = tmp;
//    thrust::transform(thrust::device, bodies, bodies + n, com_mass_+1, device_extract_com());
//    thrust::inclusive_scan(thrust::device, com_mass_, com_mass_+n+1, com_mass_, device_combine_com());
//
//    auto *tree = new device_node[tree_depth+1];
//
//    make_device_node(tree[0], 1);
//    tree[0].start_index[0] = 0;
//    tree[0].end_index[0] = n-1;
//    tree[0].children_index[0] = 1;
//
//    for (int i = 0; i < tree_depth; i++) {
//        device_add_layer(tree, keys, i+1);
//    }
//
//    for (int i = 0; i < tree_depth; i++) {
//        free_device_node(tree[i]);
//    }
//
//    delete[] tree;
//    delete[] com_mass_;
//}

struct device_node{
    int *start_index;
    int *end_index;
    int *children_index;
    int n;
};

struct device_extract_com {
    __device__ nbody_simulation::com_mass operator()(const body &body_) const {
        return {body_.pos, body_.mass};
    }
};

struct device_combine_com {
    __device__ nbody_simulation::com_mass operator()(const nbody_simulation::com_mass &a, const nbody_simulation::com_mass &b) {
        return {(a.com * a.mass + b.com * b.mass) / (a.mass + b.mass), a.mass + b.mass};
    }
};

__device__ void make_device_node(device_node &d_node, int n) {
    d_node.start_index = new int[n];
    memset(d_node.start_index, -1, n*sizeof(int));

    d_node.end_index = new int[n];
    memset(d_node.end_index, -1, n*sizeof(int));

    d_node.children_index = new int[n];
    memset(d_node.children_index, 0, n*sizeof(int));

    d_node.n = n;
}

struct device_initialize_tree {
    template <typename Tuple>
    __device__ void operator() (Tuple t) {
        device_node &layer = thrust::get<0>(t);
        int num_bodies = thrust::get<1>(t);
        make_device_node(layer, 3);
        layer.start_index[0] = 0;
        layer.end_index[0] = num_bodies-1;
        layer.children_index[0] = 1;
        layer.children_index[1] = 0;
        layer.children_index[2] = 1;
    }
};

struct device_initialize_layer {
    template <typename Tuple>
    __device__ void operator() (Tuple t) {
        device_node &layer = thrust::get<0>(t);
        int new_num_indices = thrust::get<1>(t);
        make_device_node(layer, new_num_indices);
    }
};

struct device_children_iterator {
    template <class Tuple>
    __device__ int* operator() (Tuple t) {
        return (thrust::get<0>(t) + thrust::get<1>(t))->children_index;
    }
};

//struct device_prefix_sum_layer {
//    __device__ void operator() (int idx) {
//        device_node &layer = thrust::get<0>(t);
//        int num_bodies = thrust::get<1>(t);
//        make_device_node(layer, 1);
//        layer.start_index[0] = 0;
//        layer.end_index[0] = num_bodies-1;
//        layer.children_index[0] = 1;
//    }
//};

__device__ void free_device_node(device_node &d_node) {
    delete[] d_node.start_index;
    delete[] d_node.end_index;
    delete[] d_node.children_index;
}

//__global__ void debug_kernel(int* dat, int n) {
//    for (int i = 0; i < n; i++) {
//        printf("%d: %d\n", i, dat[i]);
//    }
//}

void nbody_simulation::barnes_hut_gpu_calculate_accelerations() {
    rect3d root_rect {get_bounding_rect()};

//    size_t free, total;
//    printf("\n");
//    hipMemGetInfo(&free,&total);
//    printf("%d MB free of total %d MB\n",free/1024/1024,total/1024/1024);

    static bool called = false;
    if (!called) {
        hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ll*1024ll*1024ll*1ll); //set limit to 1gb
    }
    called = true;

    int num_bodies = bodies.size();

    if (num_bodies < 2) {
        return;
    }

    std::vector<uint64_t> keys(num_bodies);
    std::vector<int> idxes(num_bodies);
    for (int i = 0; i < num_bodies; i++) {
        const auto &constrained_pos = convert_xyz(root_rect, bodies[i].pos);
        keys[i] = get_key(constrained_pos.x, constrained_pos.y, constrained_pos.z);
        idxes[i] = i;
    }

    thrust::device_vector<uint64_t> device_keys = keys;
    thrust::device_vector<uint64_t> device_keys_copy = device_keys;

    thrust::device_vector<body> device_bodies = bodies;
    thrust::sort_by_key(device_keys.begin(), device_keys.end(), device_bodies.begin());

    thrust::device_vector<int> device_idxes = idxes;
    thrust::sort_by_key(device_keys_copy.begin(), device_keys_copy.end(), device_idxes.begin());

    thrust::device_vector<com_mass> com_mass_(num_bodies+1, com_mass{{0, 0, 0}, 0});
    thrust::transform(device_bodies.begin(), device_bodies.end(), com_mass_.begin()+1, device_extract_com());
    thrust::inclusive_scan(com_mass_.begin(), com_mass_.end(), com_mass_.begin(), device_combine_com());

    thrust::device_vector<device_node> tree(tree_depth+1);
    thrust::constant_iterator<int> num_bodies_iterator(num_bodies-1);

    thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(tree.begin(), num_bodies_iterator)),
                     1,
                     device_initialize_tree());

    for (int i = 0; i < tree_depth; i++) {
        int stage = i + 1;
        int sec_last = i;

        int prev_n = tree[sec_last].operator device_node().n;
        int *children_index = tree[sec_last].operator device_node().children_index;

        auto device_children_index = thrust::device_pointer_cast(children_index);

        thrust::inclusive_scan(device_children_index, device_children_index+prev_n, device_children_index);
//        thrust::device_ptr<int> last_children_idx = thrust::device_new<int>(1);
//        thrust::transform(thrust::device, children_index+prev_n-1, children_index+prev_n, last_children_idx, );
        thrust::for_each_n(thrust::make_zip_iterator(thrust::make_tuple(tree.begin()+stage, device_children_index+prev_n-1)),
                           1,
                           device_initialize_layer());

//        auto prev_tree_level_it = tree.begin() + sec_last;
//        auto transformed_iterator = thrust::make_transform_iterator(prev_tree_level, );

//        // combine into one, fix function above, considering extending from unary_function, can this even be done?
//        thrust::constant_iterator<thrust::device_vector<device_node>::iterator> tree_iterator(tree.begin());
//        thrust::constant_iterator<int> sec_last_iterator(sec_last);
//        auto transform_start_iterator = thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(tree_iterator, sec_last_iterator)), device_children_iterator());
//        auto transform_end_iterator = thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(tree_iterator, sec_last_iterator)), device_children_iterator());
//        thrust::inclusive_scan(thrust::device, transform_start_iterator, transform_end_iterator, transform_start_iterator);

//        thrust::inclusive_scan(thrust::device, );
    }

//    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(tree.begin(), num_bodies_iterator)), thrust::make_zip_iterator(thrust::make_tuple(tree.begin()+1, num_bodies_iterator+1)), device_initialize_tree);



//    thrust::host_vector<uint64_t> host_keys = device_keys;



//    std::sort(keys.begin(), keys.end());

//    body *device_bodies;
//    hipHostMalloc(&device_bodies, num_bodies * sizeof(body));
//
//    hipMemcpy(device_bodies, bodies.data(), num_bodies, hipMemcpyHostToDevice);
//
////    std::cout << hipGetErrorName(err) << std::endl;
//
//    uint64_t *device_keys;
//    hipHostMalloc(&device_keys, num_bodies * sizeof(uint64_t));
//
//    hipMemcpy(device_keys, keys.data(), num_bodies, hipMemcpyHostToDevice);
//
//    glm::vec3 *device_accelerations;
//    hipHostMalloc(&device_accelerations, num_bodies * sizeof(glm::vec3));
//
////    for (int i = 0; i < num_bodies; i++) {
////        device_bodies[i] = bodies[i];
////        device_keys[i] = keys[i];
////        device_accelerations[i] = {0, 0, 0};
////    }
//
////    std::cout << "bruh" << std::endl;
////    thrust::sort_by_key(thrust::device, device_keys, device_keys + num_bodies, device_bodies);
////    std::cout << "eet" << std::endl;
////    std::cout << "here" << std::endl;
//    thrust::device_ptr<uint64_t> device_keys_ptr(device_keys);
//    thrust::device_ptr<body> device_bodies_ptr(device_bodies);
//    thrust::sort_by_key(device_keys_ptr, device_keys_ptr + num_bodies, device_bodies_ptr);
////    std::cout << "oop" << std::endl;
//    barnes_hut_gpu_function<<<1, 1>>>(device_accelerations, device_bodies, device_keys, num_bodies, tree_depth, barnes_hut_factor);
//    hipDeviceSynchronize();
////    std::cout << "beep" << std::endl;
//
//    auto kernel_err = hipGetLastError();
//
////    std::cout << "kernel: " << hipGetErrorName(kernel_err) << std::endl;
//
//    hipHostFree(device_bodies);
//    hipHostFree(device_keys);
//    hipHostFree(device_accelerations);

//    exit(0);

//    thrust::device_vector<uint64_t> device_keys = keys;
//    thrust::device_vector<body> device_bodies = bodies;
//
//    thrust::sort_by_key(device_keys.begin(), device_keys.begin() + num_bodies, device_bodies.begin());
//    thrust::device_vector<com_mass> com_mass_(num_bodies+1, com_mass{{0, 0, 0}, 0});
//    thrust::transform(device_bodies.begin(), device_bodies.end(), com_mass_.begin() + 1, device_extract_com());
//    thrust::inclusive_scan(com_mass_.begin(), com_mass_.end(), com_mass_.begin(), device_combine_com());
//
//    thrust::device_vector<device_node> tree;
//    tree.push_back(device_node{thrust::device_vector<int>(1, -1), thrust::device_vector<int>(1, -1),thrust::device_vector<int>(1, 0)});
//
//    for (int i = 0; i < tree_depth; i++) {
//        tree.push_back(device_add_layer(tree.back().operator device_node(), device_keys));
//    }
}