
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#include <chrono>
using namespace std::chrono;

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
    int chunk_size = n/(gridDim.x*blockDim.x);
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * chunk_size;
    for (int i = idx; i < idx+chunk_size; i++)
        y[i] = x[i] + y[i];
}

void addCpu(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 24;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    auto start = high_resolution_clock::now();
    add<<<16384, 128>>>(N, x, y);
    hipDeviceSynchronize();
    // addCpu(N, x, y);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
  
    // To get the value of duration use the count()
    // member function on the duration object
    std::cout << duration.count() << std::endl;

    // Wait for GPU to finish before accessing on host

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        // if (fmax(maxError, fabs(y[i] - 3.0f)) != maxError) {
        //     std::cout << fmax(maxError, fabs(y[i] - 3.0f)) << " " << i << std::endl;
        // }
        // std::cout << i << std::endl;
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}